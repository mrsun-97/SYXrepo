#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

__global__ void add_vec(int *a, int *b, int *c){
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    c[k] = a[k] + b[k];
}

int repeat(int size){
    int i, a_host[size], b_host[size], c_host[size], c_fromgpu[size];
    for(i=0;i<size;i++){
        a_host[i] = i;
        b_host[i] = i/2-1;
    }

    int *a_dev, *b_dev, *c_dev;
    hipEvent_t start, stop;
    float cputime, gputime;
    //clock_t time1, time2;

    //gpu
    int blocksize = 512;
    int blocknum = size/blocksize;
    blocknum += (blocksize*blocknum >= size) ? 0 : 1;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMalloc(&a_dev, size*sizeof(int));
    hipMalloc(&b_dev, size*sizeof(int));
    hipMalloc(&c_dev, size*sizeof(int));
    dim3 dimBlock(blocksize, 1, 1);
    dim3 dimGrid(blocknum, 1, 1);

    hipEventRecord(start, 0);
    hipMemcpy(a_dev, a_host, size*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_dev, b_host, size*sizeof(int), hipMemcpyHostToDevice);


    add_vec<<<dimGrid, dimBlock>>>(a_dev, b_dev, c_dev);
    hipEventRecord(stop, 0);
    
    hipMemcpy(c_fromgpu, c_dev, size*sizeof(int), hipMemcpyDeviceToHost);

    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gputime, start, stop);
    //cudaEventDestory(start);
    //cudaEventDestory(stop);
    hipFree(a_dev);
    hipFree(b_dev);
    hipFree(c_dev);
    //cudaThreadExit();

    //cpu
    //time1 = clock();
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    for(i=0;i<size;i++){
        c_host[i] = a_host[i]+b_host[i];
    }
    //time2 = clock();
    hipEventRecord(stop, 0);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cputime, start, stop);
    //cputime = (time2 - time1)/(float)CLOCKS_PER_SEC;

    //check
    for(i=0;i<size;i++){
        if(c_host[i] != c_fromgpu[i]){
            printf("error occurs while calculating, outputs do not match\n");
            break;
        }
    }

    printf("size: %.1E,  gpu: %f,  cpu: %f,  rate: %f\n", (float)size, gputime, cputime, cputime/gputime);
    return 1;
}

int main(){

    int arraysize[] = {100000, 200000, 1000000, 2000000, 10000000, 20000000};
    int ii;

    for(ii=0;ii<6;ii++){
        repeat(arraysize[ii]);
    }
    return 0;
}